#include "hip/hip_runtime.h"

#include "foo-share-matrix.h"

#include "foo-cuda-except.h"

#ifndef FOO_SHARE_MATRIX_BLOCK_LEN
#define FOO_SHARE_MATRIX_BLOCK_LEN 16
#endif


static __global__ void foo_share_matrix_dot_kernel(void* _C, void* _A, void* _B, int matrix_len) {
		
		double* C = (double*) _C;
		double* B = (double*) _B;
		double* A = (double*) _A;
		
		int b_row = blockIdx.y;
		int b_col = blockIdx.x;
		
		double* Cs = &(C[matrix_len*b_row*FOO_SHARE_MATRIX_BLOCK_LEN + 
		                 b_col*FOO_SHARE_MATRIX_BLOCK_LEN]);
		double Cval = 0.0;
		
		int t_row = threadIdx.y;
		int t_col = threadIdx.x;
		
		int m = 0;
		int r = 0;
		
		for (m = 0; m < (matrix_len/FOO_SHARE_MATRIX_BLOCK_LEN); m++) {
			
			double* As = &(A[matrix_len*b_row*FOO_SHARE_MATRIX_BLOCK_LEN + 
			                 m*FOO_SHARE_MATRIX_BLOCK_LEN]);
			                 
			double* Bs = &(B[matrix_len*m*FOO_SHARE_MATRIX_BLOCK_LEN + 
			                 b_col*FOO_SHARE_MATRIX_BLOCK_LEN]); 
			
			__shared__ double Ash[FOO_SHARE_MATRIX_BLOCK_LEN][FOO_SHARE_MATRIX_BLOCK_LEN];
			__shared__ double Bsh[FOO_SHARE_MATRIX_BLOCK_LEN][FOO_SHARE_MATRIX_BLOCK_LEN];
			
			Ash[t_row][t_col] = As[t_row*matrix_len + t_col];
			Bsh[t_row][t_col] = Bs[t_row*matrix_len + t_col];
			
			__syncthreads();
			
			for (r = 0; r < FOO_SHARE_MATRIX_BLOCK_LEN; r++) {
				
				Cval+= Ash[t_row][r]*Bsh[r][t_col];
			}
			
			__syncthreads();				
	    }
		
		Cs[t_row*matrix_len + t_col] = Cval;
}


foo::ShareMatrix::ShareMatrix(int matrix_len) {
	
	this->count_row = matrix_len;
	this->count_column = matrix_len;
	
	(*this).init();
	
	this->p = nullptr;
}

foo::ShareMatrix::~ShareMatrix() {
	
	hipError_t err = hipSuccess;
	
	if (this->p != nullptr) {
		
		err = hipFree(this->p);
		
		if (err != hipSuccess) {
			
			throw CUDAExcept(err);
		}
		
		this->p = nullptr;
	}

}

void foo::ShareMatrix::toDevice() {
	
	hipError_t err = hipSuccess;
	
	if (this->p == nullptr) {
		
		err = hipMalloc(&(this->p), 
				         (this->count_column)*(this->count_row)*sizeof(double));
				         
		if (err != hipSuccess) {
			
			throw CUDAExcept(err);
		}
		
		err = hipMemcpy(this->p, 
					     (void*) this->data, 
						 (this->count_column)*(this->count_row)*sizeof(double),
						  hipMemcpyHostToDevice);
	
		if (err != hipSuccess) {
			
			throw CUDAExcept(err);
		}
	}

}


void foo::ShareMatrix::toHost() {
	
	hipError_t err = hipSuccess;
	
	if (this->p != nullptr) {
		
		err = hipMemcpy((void*) this->data, 
					      this->p, 
						 (this->count_column)*(this->count_row)*sizeof(double),
						  hipMemcpyDeviceToHost);
		
		if (err != hipSuccess) {
			
			throw CUDAExcept(err);
		}
		
		err = hipFree(this->p);
		
		if (err != hipSuccess) {
			
			throw CUDAExcept(err);
		}
		
		this->p = nullptr;
	}
}


foo::ShareMatrix foo::ShareMatrix::operator*(foo::ShareMatrix& m) {
	
		
		ShareMatrix res(this->count_row);
	
		if (this->p == nullptr) {
		
			(*this).toDevice();
		}
	
		if (m.p == nullptr) {
		
			m.toDevice();
		}
	
		res.toDevice();
	
		dim3 grid_size(this->count_column/FOO_SHARE_MATRIX_BLOCK_LEN,
		               this->count_row/FOO_SHARE_MATRIX_BLOCK_LEN, 1);
		               
		dim3 block_size(FOO_SHARE_MATRIX_BLOCK_LEN, FOO_SHARE_MATRIX_BLOCK_LEN, 1);

		if (this->profiler != nullptr) {
		
			(*(this->profiler)).start();
		}
	
		foo_share_matrix_dot_kernel <<<grid_size, block_size>>>(res.p,
																this->p,
	                                                            m.p,
	                                                            res.count_row);
		//Ждем когда все потоки закончат заполнение 
		hipDeviceSynchronize();
	
		if (this->profiler != nullptr) {
		
			(*(this->profiler)).stop();
		}
	
		res.toHost();
	
		return res;
	
}
