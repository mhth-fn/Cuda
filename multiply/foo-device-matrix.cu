#include "hip/hip_runtime.h"

#include "foo-device-matrix.h"

#include "foo-cuda-except.h"

#ifndef FOO_DEVICE_MATRIX_BLOCK_SIZE
#define FOO_DEVICE_MATRIX_BLOCK_SIZE 512
#endif

#ifndef FOO_DEVICE_MATRIX_BLOCK_WIDTH
#define FOO_DEVICE_MATRIX_BLOCK_WIDTH 16
#endif

#ifndef FOO_DEVICE_MATRIX_BLOCK_HEIGHT
#define FOO_DEVICE_MATRIX_BLOCK_HEIGHT 16
#endif

#if FOO_DEVICE_MATRIX_BLOCK_WIDTH*FOO_DEVICE_MATRIX_BLOCK_HEIGHT > FOO_DEVICE_MATRIX_BLOCK_SIZE
#error GPU block size need to less or equal 512 items 
#endif

static __global__ void foo_device_matrix_dot_kernel(void* _C, void* _A, void* _B, int rowc, int colc, int rc) {
		
		int j = blockIdx.x*blockDim.x + threadIdx.x;
		int i = blockIdx.y*blockDim.y + threadIdx.y;
		int r;
		
		double* C = (double*) _C;
		double* B = (double*) _B;
		double* A = (double*) _A;
		
		if ((i < rowc) && (j < colc)) {
			
			C[i*colc + j] = 0.0f;
			
			for (r = 0; r < rc; r++) {
				
				C[i*colc + j] = C[i*colc + j] + A[i*rc + r]*B[r*colc + j];
			}
		}
}

foo::DeviceMatrix::DeviceMatrix(int row_count, int column_count) {
	
	this->count_row = row_count;
	this->count_column = column_count;
	
	(*this).init();
	
	this->p = nullptr;
}

foo::DeviceMatrix::~DeviceMatrix() {
	
	hipError_t err = hipSuccess;
	
	if (this->p != nullptr) {
		
		err = hipFree(this->p);
		
		if (err != hipSuccess) {
			
			throw CUDAExcept(err);
		}
		
		this->p = nullptr;
	}

}

void foo::DeviceMatrix::toDevice() {
	
	hipError_t err = hipSuccess;
	
	if (this->p == nullptr) {
		
		err = hipMalloc(&(this->p), 
				         (this->count_column)*(this->count_row)*sizeof(double));
				         
		if (err != hipSuccess) {
			
			throw CUDAExcept(err);
		}
		
		err = hipMemcpy(this->p, 
					     (void*) this->data, 
						 (this->count_column)*(this->count_row)*sizeof(double),
						  hipMemcpyHostToDevice);
	
		if (err != hipSuccess) {
			
			throw CUDAExcept(err);
		}
	}

}


void foo::DeviceMatrix::toHost() {
	
	hipError_t err = hipSuccess;
	
	if (this->p != nullptr) {
		
		err = hipMemcpy((void*) this->data, 
					      this->p, 
						 (this->count_column)*(this->count_row)*sizeof(double),
						  hipMemcpyDeviceToHost);
		
		if (err != hipSuccess) {
			
			throw CUDAExcept(err);
		}
		
		err = hipFree(this->p);
		
		if (err != hipSuccess) {
			
			throw CUDAExcept(err);
		}
		
		this->p = nullptr;
	}
}

foo::DeviceMatrix foo::DeviceMatrix::operator*(foo::DeviceMatrix& m) {
	
		
		DeviceMatrix res(this->count_row, m.count_column);
	
		if (this->p == nullptr) {
		
			(*this).toDevice();
		}
	
		if (m.p == nullptr) {
		
			m.toDevice();
		}
	
		res.toDevice();
	
		int block_count_x, block_count_y;

		block_count_x = res.count_column/FOO_DEVICE_MATRIX_BLOCK_WIDTH +
						((res.count_column % FOO_DEVICE_MATRIX_BLOCK_WIDTH != 0)? 1: 0);
	
		block_count_y = res.count_row/FOO_DEVICE_MATRIX_BLOCK_HEIGHT +
						((res.count_row % FOO_DEVICE_MATRIX_BLOCK_HEIGHT != 0)? 1: 0);

	
		dim3 grid_size(block_count_x, block_count_y, 1);
		dim3 block_size(FOO_DEVICE_MATRIX_BLOCK_WIDTH, FOO_DEVICE_MATRIX_BLOCK_HEIGHT, 1);
	
		if (this->profiler != nullptr) {
		
			(*(this->profiler)).start();
		}
	
		foo_device_matrix_dot_kernel <<<grid_size, block_size>>>(res.p,
																 this->p,
	                                                             m.p,
	                                                             res.count_row,
	                                                             res.count_column,
	                                                             this->count_column);
		hipDeviceSynchronize();
	
		if (this->profiler != nullptr) {
		
			(*(this->profiler)).stop();
		}
	
		res.toHost();
	
		return res;
	
}

